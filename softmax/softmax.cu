#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/torch.h>
#include <cub/block/block_reduce.cuh>

namespace py = pybind11;

struct __align__(8) DataMaxSum
{                  // update the global max and sum, store the
                   // output at max_tmp and sum_tmp
    float max_tmp; // store max
    float sum_tmp; // store sum
};
__device__ __forceinline__ DataMaxSum reduce_dms_op(DataMaxSum a,
                                                    DataMaxSum b)
{
    bool a_bigger = (a.max_tmp > b.max_tmp);
    DataMaxSum bigger = a_bigger ? a : b;
    DataMaxSum smaller = a_bigger ? b : a;
    bigger.sum_tmp = bigger.sum_tmp +
                     smaller.sum_tmp * __expf(smaller.max_tmp - bigger.max_tmp);

    return bigger;
}
template <typename T, int BLOCK_DIM>
__launch_bounds__(BLOCK_DIM) __global__ void _blockSoftmaxKernel(
    T *__restrict input, T *__restrict output, int size, int dimsize,
    int stride)
{ // if set axis = 1, inputShape=[I,J,K,S]
  // tid = i(JKS) + j(KS) + k(S) + s

    // blockDim.x = size/dimsize = IKS
    // blockIdx.x = i(KS) + k(S) + s,blockIdx.x%stride = k(S) + s

    int tid =
        blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) *
                                  dimsize; // now, tid = i(JKS) + k(S) + s;

    DataMaxSum dms_partial;
    dms_partial.max_tmp = -__FLT_MAX__;
    dms_partial.sum_tmp = 0.0f;
    DataMaxSum dms_input;
    int remain = dimsize % BLOCK_DIM;
    int step = (dimsize - remain) / BLOCK_DIM + 1; // step <= numPerThread

    if (threadIdx.x < remain)
    {
        for (int ind = 0; ind < step; ind++)
        {
            dms_input.max_tmp =
                input[tid + (threadIdx.x * step + ind) * stride];

            dms_input.sum_tmp = 1.0f;
            dms_partial =
                reduce_dms_op(dms_partial,
                              dms_input); // reduce the data to one block
        }
    }
    else
    {
        for (int ind = 0; ind < step - 1; ind++)
        {
            dms_input.max_tmp =
                input[tid + (remain * step +
                             (threadIdx.x - remain) * (step - 1) + ind) *
                                stride];

            dms_input.sum_tmp = 1.0f;
            dms_partial =
                reduce_dms_op(dms_partial,
                              dms_input); // reduce the data to one block
        }
    }

    typedef hipcub::BlockReduce<DataMaxSum, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ DataMaxSum dms_total;
    DataMaxSum dms_block =
        BlockReduce(temp_storage).Reduce(dms_partial, reduce_dms_op);
    if (threadIdx.x ==
        0)
    { // must set threadIdx.x = 0 write the output to memory
        dms_total = dms_block;
    }
    __syncthreads();
    //-----------------
    if (threadIdx.x < remain)
    {
        for (int ind = 0; ind < step; ind++)
        {

            output[tid + (threadIdx.x * step + ind) * stride] =
                __expf(static_cast<float>(
                           input[tid + (threadIdx.x * step + ind) * stride]) -
                       dms_total.max_tmp) *
                __fdividef(1.0F, dms_total.sum_tmp);
        }
    }
    else
    {
        for (int ind = 0; ind < step - 1; ind++)
        {

            output[tid +
                   (remain * step + (threadIdx.x - remain) * (step - 1) + ind) *
                       stride] =
                __expf(static_cast<float>(
                           input[tid +
                                 (remain * step +
                                  (threadIdx.x - remain) * (step - 1) + ind) *
                                     stride]) -
                       dms_total.max_tmp) *
                __fdividef(1.0F, dms_total.sum_tmp);
        }
    }
}

template <typename T, int BLOCK_DIM, int numPerThread>
__global__ void
_blockSoftmaxKernel(T *__restrict input, T *__restrict output, int size,
                    int dimsize,
                    int stride)
{ // if set axis = 1, inputShape=[I,J,K,S]
  // tid = i(JKS) + j(KS) + k(S) + s

    // blockDim.x = size/dimsize = IKS
    // blockIdx.x = i(KS) + k(S) + s,blockIdx.x%stride = k(S) + s

    int tid =
        blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) *
                                  dimsize; // now, tid = i(JKS) + k(S) + s;
    int remain = dimsize % BLOCK_DIM;
    int step = (dimsize - remain) / BLOCK_DIM + 1; // step <= numPerThread
    float dataPerThread[numPerThread];

    DataMaxSum dms_partial;
    dms_partial.max_tmp = -__FLT_MAX__;
    dms_partial.sum_tmp = 0.0f;
    DataMaxSum dms_input;
    if (threadIdx.x < remain)
    {
        for (int ind = 0; ind < step; ind++)
        {
            dataPerThread[ind] =
                input[tid + (threadIdx.x * step + ind) * stride];
            dms_input.max_tmp = dataPerThread[ind];
            dms_input.sum_tmp = 1.0f;
            dms_partial =
                reduce_dms_op(dms_partial,
                              dms_input); // reduce the data to one block
        }
    }
    else
    {
        for (int ind = 0; ind < step - 1; ind++)
        {
            dataPerThread[ind] =
                input[tid + (remain * step +
                             (threadIdx.x - remain) * (step - 1) + ind) *
                                stride];
            dms_input.max_tmp = dataPerThread[ind];
            dms_input.sum_tmp = 1.0f;
            dms_partial =
                reduce_dms_op(dms_partial,
                              dms_input); // reduce the data to one block
        }
    }

    typedef hipcub::BlockReduce<DataMaxSum, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ DataMaxSum dms_total;
    DataMaxSum dms_block =
        BlockReduce(temp_storage).Reduce(dms_partial, reduce_dms_op);
    if (threadIdx.x ==
        0)
    { // must set threadIdx.x = 0 write the output to memory
        dms_total = dms_block;
    }
    __syncthreads();
    //-----------------
    if (threadIdx.x < remain)
    {
        for (int ind = 0; ind < step; ind++)
        {
            output[tid + (threadIdx.x * step + ind) * stride] =
                __expf(dataPerThread[ind] - dms_total.max_tmp) *
                __fdividef(1.0F, dms_total.sum_tmp);
        }
    }
    else
    {
        for (int ind = 0; ind < step - 1; ind++)
        {
            output[tid +
                   (remain * step + (threadIdx.x - remain) * (step - 1) + ind) *
                       stride] =
                __expf(dataPerThread[ind] - dms_total.max_tmp) *
                __fdividef(1.0F, dms_total.sum_tmp);
        }
    }
}

template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return a + b;
    }
};

template <typename T>
struct MaxOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return max(a, b);
    }
};
template <template <typename> class ReductionOp, typename T,
          int thread_group_width>
__inline__ __device__ T WarpAllReduce(T val)
{
    for (int mask = thread_group_width / 2; mask > 0; mask /= 2)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

template <typename T, int BLOCK_DIM_x, int BLOCK_DIM_y, int numPerThreadx>
__global__ void _warpSoftmaxKernel(T *__restrict input, T *__restrict output,
                                   int size, int dimsize, int stride)
{
    int otherIdx = blockIdx.x * blockDim.y + threadIdx.y;
    int otherSize = size / dimsize;
    int tid = otherIdx % stride + (otherIdx - otherIdx % stride) * dimsize;
    float dataPerThreadx[numPerThreadx];
    if (otherIdx < otherSize)
    {

        __shared__ float max_total[BLOCK_DIM_y];
        __shared__ float sum_total[BLOCK_DIM_y];
        float max_data = -__FLT_MAX__;

        for (int ph = 0; threadIdx.x + ph * BLOCK_DIM_x < dimsize; ph++)
        {
            dataPerThreadx[ph] =
                input[tid + (threadIdx.x + ph * BLOCK_DIM_x) * stride];
            max_data = max(max_data, dataPerThreadx[ph]);
        }

        max_data = WarpAllReduce<MaxOp, float, BLOCK_DIM_x>(max_data);

        if (threadIdx.x == 0)
            max_total[threadIdx.y] = max_data;

        //--------------------------------------------
        float sum_data = 0.0f;

        for (int ph = 0; threadIdx.x + ph * BLOCK_DIM_x < dimsize; ph++)
        {
            dataPerThreadx[ph] =
                __expf(dataPerThreadx[ph] - max_total[threadIdx.y]);
            sum_data += dataPerThreadx[ph];
        }

        sum_data = WarpAllReduce<SumOp, float, BLOCK_DIM_x>(sum_data);

        if (threadIdx.x == 0)
            sum_total[threadIdx.y] = sum_data;

        //--------------------------------------------

        for (int ph = 0; threadIdx.x + ph * BLOCK_DIM_x < dimsize; ph++)
        {
            output[tid + (threadIdx.x + ph * BLOCK_DIM_x) * stride] =
                dataPerThreadx[ph] * __fdividef(1.0F, sum_total[threadIdx.y]);
        }
    }
}
void softmaxLaunch(torch::Tensor input_tensor, torch::Tensor output_tensor, int size, int dimsize, int stride)
{
    // 确保输入和输出张量都是在CUDA上
    TORCH_CHECK(input_tensor.is_cuda(), "Input tensor must be on the GPU");
    TORCH_CHECK(output_tensor.is_cuda(), "Output tensor must be on the GPU");

    float *input = input_tensor.data_ptr<float>();
    float *output = output_tensor.data_ptr<float>();
    // 计算结束以后结果会自动更新到output_tensor，不需要额外复制

    int num_blocks = size / dimsize;

    if (dimsize > 1024 * 128)
    {

        int BLOCK_DIM = 1024;
        _blockSoftmaxKernel<float, 1024>
            <<<num_blocks, BLOCK_DIM>>>(input, output, size, dimsize, stride);
    }
    else if (dimsize > 1024 * 64)
    {

        int BLOCK_DIM = 1024;
        _blockSoftmaxKernel<float, 1024, 128>
            <<<num_blocks, BLOCK_DIM>>>(input, output, size, dimsize, stride);
    }
    else if (dimsize > 1024 * 32)
    {

        int BLOCK_DIM = 1024;
        _blockSoftmaxKernel<float, 1024, 64>
            <<<num_blocks, BLOCK_DIM>>>(input, output, size, dimsize, stride);
    }
    else if (dimsize > 1024 * 16)
    {

        int BLOCK_DIM = 1024;
        _blockSoftmaxKernel<float, 1024, 32>
            <<<num_blocks, BLOCK_DIM>>>(input, output, size, dimsize, stride);
    }
    else if (dimsize > 1024 * 4)
    {

        int BLOCK_DIM = 1024;
        _blockSoftmaxKernel<float, 1024, 16>
            <<<num_blocks, BLOCK_DIM>>>(input, output, size, dimsize, stride);
    }
    else if (dimsize > 1024)
    {

        int BLOCK_DIM = 1024;
        _blockSoftmaxKernel<float, 1024, 4>
            <<<num_blocks, BLOCK_DIM>>>(input, output, size, dimsize, stride);
    }
    else if (dimsize > 31)
    {
        int BLOCK_DIM_x = 32;
        int BLOCK_DIM_y = 32;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<float, 32, 32, 32>
            <<<grid_dim, block_dim>>>(input, output, size, dimsize, stride);
    }
    else if (dimsize > 15)
    {
        int BLOCK_DIM_x = 16;
        int BLOCK_DIM_y = 64;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<float, 16, 64, 2>
            <<<grid_dim, block_dim>>>(input, output, size, dimsize, stride);
    }
    else if (dimsize > 7)
    {
        int BLOCK_DIM_x = 8;
        int BLOCK_DIM_y = 128;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<float, 8, 128, 2>
            <<<grid_dim, block_dim>>>(input, output, size, dimsize, stride);
    }
    else
    {
        int BLOCK_DIM_x = 4;
        int BLOCK_DIM_y = 256;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<float, 4, 256, 2>
            <<<grid_dim, block_dim>>>(input, output, size, dimsize, stride);
    }
    hipDeviceSynchronize();
}
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    // 第一个参数"softmax"表示注册到python模块中的函数名称，可以替换为其他名字，使用方法为：模块.softmax
    // 第二个参数softmaxLaunch是上面编写的kernel launch 函数，这里需要获得该函数的地址
    // 第三个参数"Cuda Core softmax function"是描述性文字，可以修改
    // 后面的py::arg是用来为softmax定义参数的，这些参数的数目，顺序必须和softmaxLaunch保持一致，为了增加可读性，最好名字也一致
    m.def("softmax", &softmaxLaunch, "Cuda Core softmax function",
          py::arg("input"), py::arg("output"), py::arg("size"), py::arg("dimsize"), py::arg("stride"));
}
